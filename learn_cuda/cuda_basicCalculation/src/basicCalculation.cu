#include "hip/hip_runtime.h"
#include "basicCalculation.hpp"

void sumArraysCPU(float * a,float * b,float * res,const int size) {
  for(int i = 0; i < size; i += 4) {
    res[i] = a[i] + b[i];
    res[i+1] = a[i+1] + b[i+1];
    res[i+2] = a[i+2] + b[i+2];
    res[i+3] = a[i+3] + b[i+3];
  }
}

__global__ void sumArrayGPU(float* a,float* b,float* res)
{
  // printf("blockIdx: (%d, %d) threadIdx: (%d, %d)\n",
  //   blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y);
  int i = blockIdx.x * blockDim.x * 
    blockDim.y + threadIdx.x * blockDim.y + threadIdx.y;
  res[i] = a[i] + b[i];
}

void sumArraysGPU(float* a,float* b,float* res,const int size) {
  // init arrays device mem
  int dev_id = 0;
  hipSetDevice(dev_id);
  float* array1_dev = nullptr;
  float* array2_dev = nullptr;
  float* result_dev = nullptr;
  int byte_size = sizeof(float) * size;
  hipError_t status = hipMalloc(reinterpret_cast<float**>(&array1_dev), byte_size);
  spdlog::info("array1 malloc: {}", hipGetErrorString(status));
  CHECK_CUDA(status);
  status = hipMalloc(reinterpret_cast<float**>(&array2_dev), byte_size);
  spdlog::info("array2 malloc: {}", hipGetErrorString(status));
  CHECK_CUDA(status);
  status = hipMalloc(reinterpret_cast<float**>(&result_dev), byte_size);
  spdlog::info("result malloc: {}", hipGetErrorString(status));
  CHECK_CUDA(status);

  // move data host2device
  status = hipMemcpy(array1_dev, a, byte_size, hipMemcpyHostToDevice);
  spdlog::info("array1 s2d: {}", hipGetErrorString(status));
  status = hipMemcpy(array2_dev, b, byte_size, hipMemcpyHostToDevice);
  spdlog::info("array2 s2d: {}", hipGetErrorString(status));

  // create kernel function
  dim3 block(4, 4);
  dim3 grid(size / (block.x * block.y));
  sumArrayGPU<<<grid, block>>>(array1_dev, array2_dev, result_dev);

  // move data decvice2host
  status = hipMemcpy(res, result_dev, byte_size, hipMemcpyDeviceToHost);
  spdlog::info("result d2s: {}", hipGetErrorString(status));

  // free device mem
  status = hipFree(array1_dev);
  spdlog::info("free array1: {}", hipGetErrorString(status));
  status = hipFree(array2_dev);
  spdlog::info("free array2: {}", hipGetErrorString(status));
  status = hipFree(result_dev);
  spdlog::info("free result: {}", hipGetErrorString(status));
}